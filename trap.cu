#include "hip/hip_runtime.h"
/* File:     trap.cu
 * Purpose:  Implement the trapezoidal on a gpu using cuda.  This version
 *           uses a binary tree reduction in which we attempt to reduce
 *           thread divergence.  It also uses shared memory to store 
 *           intermediate results.  Assumes both threads_per_block and 
 *           blocks_per_grid are powers of 2.
 *
 * Compile:  nvcc  -arch=sm_21 -o trap trap.cu 
 * Run:      ./trap <n> <a> <b> <blocks> <threads_per_block>
 *              n is the number of trapezoids
 *              a is the left endpoint
 *              b is the right endpoint
 *
 * Input:    None
 * Output:   Result of trapezoidal applied to f(x).
 *
 * Notes:
 * 1.  The function f(x) = x^2 + 1 is hardwired
 */
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"

#define MAX_BLOCK_SZ 512

/*-------------------------------------------------------------------
 * Function:    Dev_f
 * Purpose:     The function we're integrating
 * In arg:      x
 */
__device__ float Dev_f(float x) {
   return x*x + 1;
}  /* Dev_f */

/*-------------------------------------------------------------------
 * Function:    f
 * Purpose:     The function we're integrating
 * In arg:      x
 */
float f(float x) {
   return x*x + 1;
}  /* f */



/*-------------------------------------------------------------------
 * Function:    Dev_trap  (kernel)
 * Purpose:     Implement the trapezoidal rule
 * In args:     a, b, h, n
 * Out arg:     z
 *
 */
__global__ void Dev_trap(float a, float b, float h, int n, float z[]) {
   /* Use tmp to store each thread's trapezoid area */
   /* Can't use variable dimension here             */
   __shared__ float tmp[MAX_BLOCK_SZ];
   int t = blockDim.x * blockIdx.x + threadIdx.x;
   int loc_t = threadIdx.x;
   float my_a = a + t*h;
   
   if (t < n) tmp[loc_t] = 0.5*h*(Dev_f(my_a) + Dev_f(my_a+h));
   __syncthreads();

   /* This uses a tree structure to do the additions */
   for (int stride = blockDim.x/2; stride >  0; stride /= 2) {
      if (loc_t < stride)
         tmp[loc_t] += tmp[loc_t + stride];
      __syncthreads();
   }

   /* Store the result from this cache block in z[blockIdx.x] */
   if (threadIdx.x == 0) z[blockIdx.x] = tmp[0];
}  /* Dev_trap */    


/*-------------------------------------------------------------------
 * Host code 
 */
void Get_args(int argc, char* argv[], int* n_p, float* a_p, float* b_p,
      int* threads_per_block_p, int* blocks_p);
float Serial_trap(float a, float b, int n);
float Trap_wrapper(float a, float b, int n, float z_d[],
      int blocks, int threads);


/*-------------------------------------------------------------------
 * main
 */
int main(int argc, char* argv[]) {
   int n, threads_per_block, blocks;
   float a, b, *z_d, trap;
   double start, finish;  /* Only used on host */

   Get_args(argc, argv, &n, &a, &b, &threads_per_block, &blocks);
   hipMalloc(&z_d, blocks*sizeof(float));

   GET_TIME(start);
   trap = Trap_wrapper(a, b, n, z_d, blocks, threads_per_block);
   GET_TIME(finish);

   printf("The area as computed by cuda is: %e\n", trap);
   printf("Elapsed time for cuda = %e seconds\n", finish-start);

   GET_TIME(start)
   trap = Serial_trap(a, b, n);
   GET_TIME(finish);
   printf("The area as computed by cpu is: %e\n", trap);
   printf("Elapsed time for cpu = %e seconds\n", finish-start);

   hipFree(z_d);

   return 0;
}  /* main */


/*-------------------------------------------------------------------
 * Function:  Get_args
 * Purpose:   Get and check command line args.  If there's an error
 *            quit.
 */
void Get_args(int argc, char* argv[], int* n_p, float* a_p, float* b_p,
      int* threads_per_block_p, int* blocks_p) {

   if (argc != 6) {
      fprintf(stderr, "usage: %s <n> <a> <b> <blocks> <threads per block>\n", 
            argv[0]);
      exit(0);
   }
   *n_p = strtol(argv[1], NULL, 10);
   *a_p = strtod(argv[2], NULL);
   *b_p = strtod(argv[3], NULL);
   *blocks_p = strtol(argv[4], NULL, 10);
   *threads_per_block_p = strtol(argv[5], NULL, 10);
}  /* Get_args */


/*-------------------------------------------------------------------
 * Function:  Trap_wrapper
 * Purpose:   CPU wrapper function for GPU trapezoidal rule
 * Note:      Assumes z_d has been allocated.
 */
float Trap_wrapper(float a, float b, int n, float z_d[], 
      int blocks, int threads) {
   int i;
   float trap = 0.0, h;
   float z_h[blocks];

   /* Invoke kernel */
   h = (b-a)/n;
   Dev_trap<<<blocks, threads>>>(a, b, h, n, z_d);
   hipDeviceSynchronize();

   hipMemcpy(&z_h, z_d, blocks*sizeof(float), hipMemcpyDeviceToHost);

   for (i = 0; i < blocks; i++)
      trap += z_h[i];
   return trap;
}  /* Trap_wrapper */


/*-------------------------------------------------------------------
 * Function:  Serial_dot
 * Purpose:   Compute a dot product on the cpu
 */
float Serial_trap(float a, float b, int n) {
   int i;
   float x, h, trap = 0;

   h = (b-a)/n;

   trap = (f(a) + f(b))/2.0;
   for (i = 1; i <= n-1; i++) {
       x = a + i*h;
       trap = trap + f(x);
   }
   trap = trap*h;
   
   return trap;
}  /* Serial_trap */
